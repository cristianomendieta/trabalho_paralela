#include "hip/hip_runtime.h"
/**
 * CI1009 - PROGRAMAÇÃO PARALELA COM GPUs
 * 2º semestre de 2025
 * UFPR - Prof. W.Zola
 * Autores: Cristiano Creppo Mendieta e Thiago Ruiz
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include "chrono.c"

#define BLOCK_SIZE 1024

__global__ void copyKernel(float* output, float* input, unsigned int n) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        output[i] = input[i];
    }
}

int main(int argc, char* argv[]) {
    if (argc != 2) {
        printf("Usage: %s <nElements>\n", argv[0]);
        exit(1);
    }
    
    unsigned int nElements = atol(argv[1]);
    int NTIMES = 30;
    
    printf("=== copyKernel - Medição de Largura de Banda ===\n");
    printf("Elementos: %u\n", nElements);
    printf("Repetições: %d\n\n", NTIMES);
    
    int dev = 0;
    hipSetDevice(dev);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("GPU: %s\n\n", deviceProp.name);
    
    size_t size = nElements * sizeof(float);
    float* input_h = (float*)malloc(size);
    float* output_h = (float*)malloc(size);
    
    for (unsigned int i = 0; i < nElements; i++) {
        input_h[i] = (float)i;
    }
    
    float *input_d, *output_d;
    checkCudaErrors(hipMalloc(&input_d, size));
    checkCudaErrors(hipMalloc(&output_d, size));
    checkCudaErrors(hipMemcpy(input_d, input_h, size, hipMemcpyHostToDevice));
    
    int threadsPerBlock = BLOCK_SIZE;
    int blocksPerGrid = (nElements + threadsPerBlock - 1) / threadsPerBlock;
    
    printf("Configuração: %d blocos × %d threads\n", blocksPerGrid, threadsPerBlock);
    
    copyKernel<<<blocksPerGrid, threadsPerBlock>>>(output_d, input_d, nElements);
    hipDeviceSynchronize();
    
    chronometer_t chrono;
    chrono_reset(&chrono);
    chrono_start(&chrono);
    
    for (int i = 0; i < NTIMES; i++) {
        copyKernel<<<blocksPerGrid, threadsPerBlock>>>(output_d, input_d, nElements);
    }
    hipDeviceSynchronize();
    chrono_stop(&chrono);
    
    checkCudaErrors(hipMemcpy(output_h, output_d, size, hipMemcpyDeviceToHost));
    
    bool correct = true;
    for (unsigned int i = 0; i < nElements && correct; i++) {
        if (output_h[i] != input_h[i]) {
            correct = false;
        }
    }
    
    printf("\n=== RESULTADOS ===\n");
    printf("Validação: %s\n", correct ? "CORRETO" : "INCORRETO");
    
    chrono_report_TimeInLoop(&chrono, "copyKernel", NTIMES);
    
    double total_time_s = chrono_gettotal(&chrono) / 1e9;
    double bytes_transferred = 2.0 * nElements * sizeof(float) * NTIMES;
    double bandwidth_GBs = bytes_transferred / total_time_s / 1e9;
    
    double throughput_GFLOPS = ((double)nElements * NTIMES) / chrono_gettotal(&chrono);
    
    printf("\n=== DESEMPENHO ===\n");
    printf("Largura de Banda: %.2f GB/s\n", bandwidth_GBs);
    printf("Throughput: %.2f GFLOPS\n", throughput_GFLOPS);
    
    double theoretical_bandwidth = 86.4;
    double efficiency = (bandwidth_GBs / theoretical_bandwidth) * 100.0;
    printf("Eficiência: %.1f%% da largura de banda teórica (%.1f GB/s)\n", 
           efficiency, theoretical_bandwidth);
    
    free(input_h);
    free(output_h);
    checkCudaErrors(hipFree(input_d));
    checkCudaErrors(hipFree(output_d));
    
    printf("\nDone\n");
    return 0;
}
