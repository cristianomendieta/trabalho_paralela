#include "hip/hip_runtime.h"
/**
 * copyKernel - Kernel simples de cópia para medir largura de banda
 * 
 * Este kernel serve como baseline para comparar o desempenho dos
 * kernels de redução. Ele mede a largura de banda máxima da GPU
 * ao fazer a operação mais simples: copiar dados.
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include "hip/hip_runtime_api.h"
#include "chrono.c"

#define BLOCK_SIZE 1024

// Kernel de cópia simples
__global__ void copyKernel(float* output, float* input, unsigned int n) {
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        output[i] = input[i];
    }
}

int main(int argc, char* argv[]) {
    if (argc != 2) {
        printf("Usage: %s <nElements>\n", argv[0]);
        exit(1);
    }
    
    unsigned int nElements = atol(argv[1]);
    int NTIMES = 30;
    
    printf("=== copyKernel - Medição de Largura de Banda ===\n");
    printf("Elementos: %u\n", nElements);
    printf("Repetições: %d\n\n", NTIMES);
    
    // Configurar GPU
    int dev = 0;
    hipSetDevice(dev);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    printf("GPU: %s\n\n", deviceProp.name);
    
    // Alocar memória
    size_t size = nElements * sizeof(float);
    float* input_h = (float*)malloc(size);
    float* output_h = (float*)malloc(size);
    
    // Inicializar dados
    for (unsigned int i = 0; i < nElements; i++) {
        input_h[i] = (float)i;
    }
    
    // Alocar na GPU
    float *input_d, *output_d;
    checkCudaErrors(hipMalloc(&input_d, size));
    checkCudaErrors(hipMalloc(&output_d, size));
    checkCudaErrors(hipMemcpy(input_d, input_h, size, hipMemcpyHostToDevice));
    
    // Configurar kernel
    int threadsPerBlock = BLOCK_SIZE;
    int blocksPerGrid = (nElements + threadsPerBlock - 1) / threadsPerBlock;
    
    printf("Configuração: %d blocos × %d threads\n", blocksPerGrid, threadsPerBlock);
    
    // Warm up
    copyKernel<<<blocksPerGrid, threadsPerBlock>>>(output_d, input_d, nElements);
    hipDeviceSynchronize();
    
    // Medir tempo
    chronometer_t chrono;
    chrono_reset(&chrono);
    chrono_start(&chrono);
    
    for (int i = 0; i < NTIMES; i++) {
        copyKernel<<<blocksPerGrid, threadsPerBlock>>>(output_d, input_d, nElements);
    }
    hipDeviceSynchronize();
    chrono_stop(&chrono);
    
    // Copiar resultado de volta para validar
    checkCudaErrors(hipMemcpy(output_h, output_d, size, hipMemcpyDeviceToHost));
    
    // Validar
    bool correct = true;
    for (unsigned int i = 0; i < nElements && correct; i++) {
        if (output_h[i] != input_h[i]) {
            correct = false;
        }
    }
    
    // Reportar resultados
    printf("\n=== RESULTADOS ===\n");
    printf("Validação: %s\n", correct ? "CORRETO" : "INCORRETO");
    
    chrono_report_TimeInLoop(&chrono, "copyKernel", NTIMES);
    
    // Calcular largura de banda
    double total_time_s = chrono_gettotal(&chrono) / 1e9; // converter para segundos
    double bytes_transferred = 2.0 * nElements * sizeof(float) * NTIMES; // leitura + escrita
    double bandwidth_GBs = bytes_transferred / total_time_s / 1e9; // GB/s
    
    // Calcular vazão em GFLOPS (elementos/segundo convertido)
    double throughput_GFLOPS = ((double)nElements * NTIMES) / chrono_gettotal(&chrono);
    
    printf("\n=== DESEMPENHO ===\n");
    printf("Largura de Banda: %.2f GB/s\n", bandwidth_GBs);
    printf("Throughput: %.2f GFLOPS\n", throughput_GFLOPS);
    
    // Comparar com largura de banda teórica da GPU
    // GTX 750 Ti: 86.4 GB/s teórico
    double theoretical_bandwidth = 86.4; // GB/s para GTX 750 Ti
    double efficiency = (bandwidth_GBs / theoretical_bandwidth) * 100.0;
    printf("Eficiência: %.1f%% da largura de banda teórica (%.1f GB/s)\n", 
           efficiency, theoretical_bandwidth);
    
    // Limpeza
    free(input_h);
    free(output_h);
    checkCudaErrors(hipFree(input_d));
    checkCudaErrors(hipFree(output_d));
    
    printf("\nDone\n");
    return 0;
}
