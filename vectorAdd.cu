#include "hip/hip_runtime.h"
// vectprAdd.cu 
// Alterado por W. Zola (ago 2025)
// para uso em ci1009

/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
//#include <hip/hip_runtime.h>  // comentado por WZ (não precisamos disso!)

//#include <hip/hip_runtime_api.h>
#include "hip/hip_runtime_api.h"   // mudado aqui por WZ
#include <assert.h>
#include "chrono.c"

chronometer_t c1;      // declare a chronometer (you could have more!)
int NTIMES = 1; //100;     // NTIMES to repeat our experiments
       
/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void
vectorAdd(const float *A, const float *B, float *C, int numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = A[i] + B[i];
    }
}

/**
 * Host main routine
 */
int
main(int argc, char *argv[])
{
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    long numElements = 1*1000*1000;   // 1 milhao (default)
    size_t size = numElements * sizeof(float);
    
    if( argc != 3 && argc != 2 ) {
	   printf( "Usage: %s <nElements> [<nBlocks>] \n", argv[0] );
	   printf( "Note that the number of blocks is optional\n" );
           printf( "If nBlocks is specified the persistent kernel is used\n" );
           printf( "If nBlocks is omitted the many-threads kernel is used\n" );
	   exit( 0 ); 
    }

    int usePersistentKernel = 0, useManyThreadsKernel = 0;

    if( argc == 3 ) {
       usePersistentKernel = 1;
       printf( "Using persistent kernel\n" );
       
    } else if( argc == 2 ) {
       useManyThreadsKernel = 1;
       printf( "Using ManyThreads kernel\n" );
    }
    
    numElements = atol(argv[1]);
    printf( "Number of elements requested in command line: %ld \n", 
             numElements );
    size = numElements * sizeof(float);   // size of arrays (in bytes)         

    int dev = 0;            // wz: assume only one GPU for simplicity!
    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);
    if (error_id != hipSuccess) {
      printf("hipGetDeviceCount returned %d\n-> %s\n",
           static_cast<int>(error_id), hipGetErrorString(error_id));
      printf("Result = FAIL\n");
      exit(EXIT_FAILURE);
    }
  
    hipSetDevice(dev);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    
    printf("\nGPU Device %d name is \"%s\"\n", dev, deviceProp.name);
    


    
    // Allocate the host input vector A
    float *h_A = (float *)malloc(size);

    // Allocate the host input vector B
    float *h_B = (float *)malloc(size);

    // Allocate the host output vector C
    float *h_C = (float *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // Allocate the device input vector A
    float *d_A = NULL;
    err = hipMalloc((void **)&d_A, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, 
                 "Failed to allocate device vector A (error code %s)!\n", 
                 hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector B
    float *d_B = NULL;
    err = hipMalloc((void **)&d_B, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, 
                 "Failed to allocate device vector B (error code %s)!\n", 
                 hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    float *d_C = NULL;
    err = hipMalloc((void **)&d_C, size);

    if (err != hipSuccess)
    {
        fprintf(stderr, 
                 "Failed to allocate device vector C (error code %s)!\n", 
                 hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, 
                 "Failed to copy vector A from host to device (error code %s)!\n", 
                 hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    if (err != hipSuccess)
    {
        fprintf(stderr, 
                 "Failed to copy vector B from host to device (error code %s)!\n", 
                 hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    int blocksPerGrid;    // number of thread blocks
    if( usePersistentKernel ) {
    
       blocksPerGrid = atoi(argv[2]);   // get number of 
                                   //  persistent thread blocks from command line
       fprintf(stderr, 
                 "codigo da versao persistente NAO feito ainda!\n" );                             
        
    } else {

       // use NVIDIA regular (manythreads) model
       assert( useManyThreadsKernel == 1 );
       // Launch the Vector Add CUDA Kernel
       int threadsPerBlock = 768;     // wz: mudar AQUI para ser o maximo na sua GPU
       blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
       printf("CUDA kernel launch with %d blocks of %d threads\n", 
               blocksPerGrid, threadsPerBlock);
               
       chrono_reset(&c1);

       chrono_reset(&c1);

       vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, 1);  // WARM UP
       // usar hipDeviceSynchronize() para esperar término de 
       //   hipMemcpy e/ou warm-up kernels iniciados anteriormente
       //   ANTES de ligar o cronometro
       hipDeviceSynchronize();   
       chrono_start(&c1);
         for (int i = 0; i < NTIMES; i++) {

           
               
           vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);

           err = hipGetLastError();

           if (err != hipSuccess)
           {
               fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", 
                                hipGetErrorString(err));
               exit(EXIT_FAILURE);
           }
         }
         hipDeviceSynchronize();
       chrono_stop(&c1);

    }
    
    // Cálculo da VAZAO
    
      printf("\nGPU: %s vectorAdd kernel\n", 
             deviceProp.name );
      chrono_report_TimeInLoop(&c1, "CUDA kernel launch", NTIMES);
      printf("vectorAdd Throughput: %lf floats/ns (or Giga FLOPS, in this case)\n", 
                ((double)numElements*NTIMES)/((double)chrono_gettotal(&c1)));
                
      // OBS: veja que SAO lidos 2 vetores (A e B) e produzido um terceiro vetor C
      //      entao para calculo da vazao de acesso à memoria devemos 
      //      multiplicar (abaixo) o numero de elementos por 3         
      printf("Global Memory Throughput: %lf GiB/s (Giga Bytes/s)\n", 
                ((double)sizeof(uint32_t)*numElements*3*NTIMES) / ((double)chrono_gettotal(&c1)) );
          
    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    if (err != hipSuccess)
    {
        fprintf(stderr, 
                 "Failed to copy vector C from device to host (error code %s)!\n", 
                 hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(h_A[i] + h_B[i] - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }

    printf("Test PASSED\n");

    // Free device global memory
    err = hipFree(d_A);

    if (err != hipSuccess)
    {
        fprintf(stderr, 
                 "Failed to free device vector A (error code %s)!\n", 
                 hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_B);

    if (err != hipSuccess)
    {
        fprintf(stderr, 
                 "Failed to free device vector B (error code %s)!\n", 
                 hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipFree(d_C);

    if (err != hipSuccess)
    {
        fprintf(stderr, 
                 "Failed to free device vector C (error code %s)!\n", 
                 hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    printf("Done\n");
    return 0;
}

