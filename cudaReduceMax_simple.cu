#include "hip/hip_runtime.h"
/**
 * CI1009 - PROGRAMAÇÃO PARALELA COM GPUs
 * 2º semestre de 2025
 * UFPR - Prof. W.Zola
 * 
 * Trabalho 1: Versões paralela CUDA para kernels persistentes do algoritmo de redução
 * 
 * Implementação de kernels de redução paralela para encontrar o valor máximo
 * de um vetor de números float usando CUDA.
 * 
 * VERSÃO SIMPLIFICADA - SEM THRUST (para evitar problemas de linkagem)
 * 
 * Autor: [Seu Nome]
 * Data: Outubro 2025
 */

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <math.h>
#include <time.h>
#include <assert.h>

// Incluir arquivos auxiliares
#include "hip/hip_runtime_api.h"
#include "chrono.c"

#define BLOCK_SIZE 1024
#define MAX_BLOCKS 65535

// Chronometer global
chronometer_t chrono_kernel1, chrono_kernel2;
int NTIMES = 30; // Número padrão de repetições

// Função atomicMax para float (usando int como base)
__device__ float atomicMax(float* address, float val) {
    int* address_as_i = (int*) address;
    int old = *address_as_i, assumed;
    do {
        assumed = old;
        old = atomicCAS(address_as_i, assumed,
            __float_as_int(fmaxf(val, __int_as_float(assumed))));
    } while (assumed != old);
    return __int_as_float(old);
}

// Função para exibir uso do programa
void usage(const char* progname) {
    printf("Usage: %s <nTotalElements> [<nBlocks>]\n", progname);
    printf("  nTotalElements: número de floats do vetor de entrada\n");
    printf("  nBlocks: número de blocos (opcional - usa kernel persistente)\n");
    printf("Note que o número de blocos é opcional\n");
    printf("Se nBlocks é especificado, o kernel persistente é usado\n");
    printf("Se nBlocks é omitido, o kernel many-threads é usado\n");
    exit(1);
}

// Função para gerar números aleatórios conforme especificação
void generateInput(float* Input_h, unsigned int nElements) {
    srand(time(NULL));
    for (unsigned int i = 0; i < nElements; i++) {
        int a = rand();
        int b = rand();
        float v = a * 100.0f + b;
        Input_h[i] = v;
    }
}

// Kernel 1: Versão many-threads com redução eficiente
__global__ void reduceMax(float* input, float* output, unsigned int n) {
    extern __shared__ float sdata[];
    
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
    
    // Cada thread carrega um elemento (ou -INFINITY se fora dos limites)
    sdata[tid] = (i < n) ? input[i] : -INFINITY;
    __syncthreads();
    
    // Redução em shared memory usando tree reduction
    for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
        if (tid < s) {
            sdata[tid] = fmaxf(sdata[tid], sdata[tid + s]);
        }
        __syncthreads();
    }
    
    // Thread 0 escreve resultado do bloco
    if (tid == 0) {
        output[blockIdx.x] = sdata[0];
    }
}

// Kernel 2: Versão persistente com atomics
__global__ void reduceMax_atomic_persist(float* max_result, float* input, unsigned int nElements) {
    extern __shared__ float shared_max[];
    
    unsigned int tid = threadIdx.x;
    unsigned int bid = blockIdx.x;
    unsigned int globalTid = bid * blockDim.x + tid;
    unsigned int gridSize = blockDim.x * gridDim.x;
    
    // Inicializar shared memory
    if (tid == 0) {
        shared_max[0] = -INFINITY;
    }
    __syncthreads();
    
    // Fase 1: Cada thread processa múltiplos elementos de forma coalescida
    float thread_max = -INFINITY;
    for (unsigned int i = globalTid; i < nElements; i += gridSize) {
        thread_max = fmaxf(thread_max, input[i]);
    }
    
    // Fase 2: Redução dentro do bloco usando atomic em shared memory
    atomicMax(&shared_max[0], thread_max);
    __syncthreads();
    
    // Fase 3: Thread 0 de cada bloco faz atomic em global memory
    if (tid == 0) {
        atomicMax(max_result, shared_max[0]);
    }
}

// Wrapper para redução completa (kernel 1)
float reduceMaxComplete(float* input_d, unsigned int nElements) {
    int threadsPerBlock = BLOCK_SIZE;
    int blocksPerGrid = (nElements + threadsPerBlock - 1) / threadsPerBlock;
    
    // Primeira passada
    float* temp_d;
    checkCudaErrors(hipMalloc(&temp_d, blocksPerGrid * sizeof(float)));
    
    reduceMax<<<blocksPerGrid, threadsPerBlock, threadsPerBlock * sizeof(float)>>>(
        input_d, temp_d, nElements);
    checkCudaErrors(hipGetLastError());
    
    // Se temos apenas um bloco, terminamos
    if (blocksPerGrid == 1) {
        float result;
        checkCudaErrors(hipMemcpy(&result, temp_d, sizeof(float), hipMemcpyDeviceToHost));
        checkCudaErrors(hipFree(temp_d));
        return result;
    }
    
    // Senão, precisamos de mais reduções
    while (blocksPerGrid > 1) {
        int newBlocksPerGrid = (blocksPerGrid + threadsPerBlock - 1) / threadsPerBlock;
        
        reduceMax<<<newBlocksPerGrid, threadsPerBlock, threadsPerBlock * sizeof(float)>>>(
            temp_d, temp_d, blocksPerGrid);
        checkCudaErrors(hipGetLastError());
        
        blocksPerGrid = newBlocksPerGrid;
    }
    
    float result;
    checkCudaErrors(hipMemcpy(&result, temp_d, sizeof(float), hipMemcpyDeviceToHost));
    checkCudaErrors(hipFree(temp_d));
    return result;
}

int main(int argc, char* argv[]) {
    // Verificar argumentos da linha de comando
    if (argc != 2 && argc != 3) {
        usage(argv[0]);
    }
    
    unsigned int nTotalElements = atol(argv[1]);
    printf("Number of elements requested in command line: %u\n", nTotalElements);
    
    if (nTotalElements <= 0) {
        fprintf(stderr, "Erro: número de elementos deve ser positivo\n");
        exit(1);
    }
    
    // Determinar qual kernel usar
    int usePersistentKernel = 0, useManyThreadsKernel = 0;
    int nBlocks = 0;
    
    if (argc == 3) {
        usePersistentKernel = 1;
        nBlocks = atoi(argv[2]);
        printf("Using persistent kernel with %d blocks\n", nBlocks);
    } else {
        useManyThreadsKernel = 1;
        printf("Using ManyThreads kernel\n");
    }
    
    printf("=== CUDA Reduce Max - Kernels Persistentes ===\n");
    printf("Elementos: %u\n", nTotalElements);
    printf("Repetições: %d\n\n", NTIMES);
    
    // Configuração da GPU
    int dev = 0;
    int deviceCount = 0;
    hipError_t error_id = hipGetDeviceCount(&deviceCount);
    if (error_id != hipSuccess) {
        printf("hipGetDeviceCount returned %d\n-> %s\n",
               static_cast<int>(error_id), hipGetErrorString(error_id));
        printf("Result = FAIL\n");
        exit(EXIT_FAILURE);
    }
    
    hipSetDevice(dev);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, dev);
    
    printf("\nGPU Device %d name is \"%s\"\n", dev, deviceProp.name);
    
    // Alocar memória no host
    size_t size = nTotalElements * sizeof(float);
    float* Input_h = (float*)malloc(size);
    if (!Input_h) {
        fprintf(stderr, "Erro: falha na alocação de memória no host\n");
        exit(EXIT_FAILURE);
    }
    
    // Gerar dados de entrada
    printf("Gerando dados de entrada...\n");
    generateInput(Input_h, nTotalElements);
    
    // Alocar memória na GPU
    float* Input_d = NULL;
    float* result_d = NULL;
    checkCudaErrors(hipMalloc(&Input_d, size));
    checkCudaErrors(hipMalloc(&result_d, sizeof(float)));
    
    // Copiar dados para GPU
    printf("Copy input data from the host memory to the CUDA device\n");
    checkCudaErrors(hipMemcpy(Input_d, Input_h, size, hipMemcpyHostToDevice));
    
    // Calcular máximo na CPU para validação
    float max_cpu = Input_h[0];
    for (unsigned int i = 1; i < nTotalElements; i++) {
        if (Input_h[i] > max_cpu) {
            max_cpu = Input_h[i];
        }
    }
    printf("Máximo calculado na CPU: %f\n", max_cpu);
    
    float result_kernel1 = 0.0f, result_kernel2 = 0.0f;
    
    printf("\n=== TESTES DOS KERNELS ===\n");
    
    // Teste Kernel 1 (Many-threads) se selecionado
    if (useManyThreadsKernel) {
        printf("Testando Kernel 1 (Many-threads)...\n");
        
        int threadsPerBlock = BLOCK_SIZE;
        int blocksPerGrid = (nTotalElements + threadsPerBlock - 1) / threadsPerBlock;
        printf("CUDA kernel launch with %d blocks of %d threads\n", 
               blocksPerGrid, threadsPerBlock);
        
        chrono_reset(&chrono_kernel1);
        
        // Warm up
        result_kernel1 = reduceMaxComplete(Input_d, nTotalElements);
        hipDeviceSynchronize();
        
        chrono_start(&chrono_kernel1);
        for (int i = 0; i < NTIMES; i++) {
            result_kernel1 = reduceMaxComplete(Input_d, nTotalElements);
        }
        hipDeviceSynchronize();
        chrono_stop(&chrono_kernel1);
    }
    
    // Teste Kernel 2 (Persistente) se selecionado
    if (usePersistentKernel) {
        printf("Testando Kernel 2 (Persistente) com %d blocos...\n", nBlocks);
        
        chrono_reset(&chrono_kernel2);
        
        // Warm up
        float init_val = -INFINITY;
        checkCudaErrors(hipMemcpy(result_d, &init_val, sizeof(float), hipMemcpyHostToDevice));
        reduceMax_atomic_persist<<<nBlocks, BLOCK_SIZE, sizeof(float)>>>(result_d, Input_d, nTotalElements);
        hipDeviceSynchronize();
        
        chrono_start(&chrono_kernel2);
        for (int i = 0; i < NTIMES; i++) {
            init_val = -INFINITY;
            checkCudaErrors(hipMemcpy(result_d, &init_val, sizeof(float), hipMemcpyHostToDevice));
            reduceMax_atomic_persist<<<nBlocks, BLOCK_SIZE, sizeof(float)>>>(result_d, Input_d, nTotalElements);
        }
        hipDeviceSynchronize();
        chrono_stop(&chrono_kernel2);
        
        // Obter resultado do kernel 2
        checkCudaErrors(hipMemcpy(&result_kernel2, result_d, sizeof(float), hipMemcpyDeviceToHost));
    }
    
    // Validar resultados
    printf("\n=== RESULTADOS ===\n");
    printf("CPU:               %f\n", max_cpu);
    if (useManyThreadsKernel) printf("Kernel 1:          %f\n", result_kernel1);
    if (usePersistentKernel) printf("Kernel 2:          %f\n", result_kernel2);
    
    // Verificar se os resultados estão corretos
    float tolerance = 1e-3f;
    bool correct1 = !useManyThreadsKernel || fabsf(result_kernel1 - max_cpu) < tolerance;
    bool correct2 = !usePersistentKernel || fabsf(result_kernel2 - max_cpu) < tolerance;
    
    printf("\nValidação:\n");
    if (useManyThreadsKernel) printf("Kernel 1: %s\n", correct1 ? "CORRETO" : "INCORRETO");
    if (usePersistentKernel) printf("Kernel 2: %s\n", correct2 ? "CORRETO" : "INCORRETO");
    
    // Reportar desempenho
    printf("\n=== DESEMPENHO ===\n");
    printf("\nGPU: %s reduceMax kernel\n", deviceProp.name);
    
    if (useManyThreadsKernel) {
        chrono_report_TimeInLoop(&chrono_kernel1, "Kernel 1 (Many-threads)", NTIMES);
        printf("reduceMax Kernel1 Throughput: %lf GFLOPS\n", 
               ((double)nTotalElements*NTIMES)/((double)chrono_gettotal(&chrono_kernel1)));
    }
    
    if (usePersistentKernel) {
        chrono_report_TimeInLoop(&chrono_kernel2, "Kernel 2 (Persistente)", NTIMES);
        printf("reduceMax Kernel2 Throughput: %lf GFLOPS\n", 
               ((double)nTotalElements*NTIMES)/((double)chrono_gettotal(&chrono_kernel2)));
    }
    
    // Calcular acelerações entre kernels se ambos foram executados
    if (useManyThreadsKernel && usePersistentKernel) {
        double speedup = (double)chrono_gettotal(&chrono_kernel1) / (double)chrono_gettotal(&chrono_kernel2);
        printf("Aceleração Kernel2 vs Kernel1: %.2fx\n", speedup);
    }
    
    printf("Test PASSED\n");
    
    // Limpeza
    free(Input_h);
    checkCudaErrors(hipFree(Input_d));
    checkCudaErrors(hipFree(result_d));
    
    printf("\nDone\n");
    return 0;
}